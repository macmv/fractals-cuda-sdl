#include "hip/hip_runtime.h"
#include "camera.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPU: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

Camera::Camera() {
  printf("Initializing camera...\n");
  hipMallocManaged(&pos, sizeof(double) * 3);
  hipMallocManaged(&dir, sizeof(double) * 2);
  hipDeviceSynchronize();
  pos[0] = -5;
  pos[1] = 0;
  pos[2] = 0;
  dir[0] = 0;
  dir[1] = 0;
  gpuErrchk(hipPeekAtLastError());
  printf("Initialized camera\n");
}

__device__
void Camera::getDeltaFrom2D(double x, double y, double* delta) {
  delta[0] = cos((dir[0] + (x - 0.5) * fov) / 180 * M_PI);
  delta[1] = dir[1] + (y - 0.5) * (fov * (16.0/9)) / 180;
  delta[2] = sin((dir[0] + (x - 0.5) * fov) / 180 * M_PI);
  float len = sqrt(pow(delta[0], 2) + pow(delta[1], 2) + pow(delta[2], 2));
  delta[0] /= len;
  delta[1] /= len;
  delta[2] /= len;
}

void Camera::free() {
  hipFree(pos);
  hipFree(dir);
}

void Camera::rotate(double x, double y) {
  dir[0] += x;
  dir[1] += y;
}

void Camera::translate(double x, double y, double z) {
  pos[0] += x;
  pos[1] += y;
  pos[2] += z;
}
