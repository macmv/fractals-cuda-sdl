#include "hip/hip_runtime.h"
#include "camera.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPU: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

Camera::Camera() {
  printf("Initializing camera...\n");
  hipMallocManaged(&pos, sizeof(double) * 3);
  hipMallocManaged(&dir, sizeof(double) * 2);
  hipDeviceSynchronize();
  pos[0] = -5;
  pos[1] = 0;
  pos[2] = 0;
  dir[0] = 0;
  dir[1] = 0;
  gpuErrchk(hipPeekAtLastError());
  printf("Initialized camera\n");
}

__device__
void Camera::getDeltaFrom2D(double x, double y, double* delta) {
  delta[0] = cos((dir[0] + (x - 0.5) * fov) / 180 * M_PI);
  delta[1] = dir[1] + (y - 0.5) * (fov * (16.0/9)) / 180;
  delta[2] = sin((dir[0] + (x - 0.5) * fov) / 180 * M_PI);
}

void Camera::free() {
  hipFree(pos);
  hipFree(dir);
}
