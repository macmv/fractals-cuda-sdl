#include "hip/hip_runtime.h"
#include "camera.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPU: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

Camera::Camera() {
  printf("Initializing camera...\n");
  hipMallocManaged(&pos, sizeof(double) * 3);
  hipMallocManaged(&dir, sizeof(double) * 2);
  hipDeviceSynchronize();
  pos[0] = -5;
  pos[1] = 0;
  pos[2] = 0;
  dir[0] = 0;
  dir[1] = 0;
  gpuErrchk(hipPeekAtLastError());
  printf("Initialized camera\n");
}

__device__
void Camera::getDeltaFrom2D(double x, double y, double* delta) {
  delta[0] = cos((dir[0] + (x - 0.5) * xFov) * 2 * M_PI); // should not nned to be normalized if my math is right
  delta[1] = sin((dir[0] + (y - 0.5) * xFov) * 2 * M_PI);
  delta[2] = 0;
}

void Camera::free() {
  hipFree(pos);
  hipFree(dir);
}
