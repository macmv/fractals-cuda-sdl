#include "camera.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPU: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}

Camera::Camera() {
  printf("Initializing camera...\n");
  hipMalloc(&pos, sizeof(pos[0]) * 3);
  hipMalloc(&dir, sizeof(pos[0]) * 2);
  hipDeviceSynchronize();
  gpuErrchk(hipPeekAtLastError());
  printf("Initialized camera\n");
}

__device__
void Camera::getDeltaFrom2D(double x, double y, double* delta) {
  delta[0] = x;
  delta[1] = y;
  delta[2] = pos[0];
}

void Camera::free() {
  hipFree(pos);
  hipFree(dir);
}
